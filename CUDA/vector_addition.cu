// vector_addition.cu
#include <stdio.h>
#include <hip/hip_runtime.h>

// CUDA kernel for vector addition
__global__ void vectorAdd(const float *A, const float *B, float *C, int N) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}

int main() {
    int N = 1 << 20; // 1 Million elements
    size_t size = N * sizeof(float);

    // Allocate host memory
    float *h_A = (float*)malloc(size);
    float *h_B = (float*)malloc(size);
    float *h_C = (float*)malloc(size);

    // Initialize host arrays
    for(int i = 0; i < N; i++) {
        h_A[i] = static_cast<float>(i);
        h_B[i] = static_cast<float>(i * 2);
    }

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // Copy host data to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Launch kernel with 256 threads per block
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Copy result back to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Verify the result
    bool success = true;
    for(int i = 0; i < N; i++) {
        if(h_C[i] != h_A[i] + h_B[i]) {
            printf("Mismatch at index %d: %f != %f\n", i, h_C[i], h_A[i] + h_B[i]);
            success = false;
            break;
        }
    }

    if(success) {
        printf("Vector addition successful!\n");
    }

    // Print the results of vector addition
    for(int i = 0; i < 10; i++) { // Print first 10 elements for brevity
        printf("C[%d] = %f\n", i, h_C[i]);
    }
    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
